
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define blocksize 64

void printMat(double *mat, int n)
{
    for (int i = 0; i < n; ++i)
    {
        for (int j = n; j < 2 * n; ++j)
        {
            printf("%lf ", mat[i * 2 * n + j]);
        }
        printf("\n");
    }
    printf("\n");
}

__global__ void swap_zero_diagonal_and_get_scale(double *mat, int i, int dim, double *scale, int *status)
{
    int col_size = 2 * dim;

    if (mat[i * col_size + 1] == 0)
    {
        for (int j = i + 1; j < dim; j++)
        {
            if (mat[j * col_size + i] != 0.0)
            {
                for (int l = 0; l < col_size; l++)
                {
                    double *row_a = &mat[i * col_size];
                    double *row_b = &mat[j * col_size];
                    double temp = row_a[l];
                    row_a[l] = row_b[l];
                    row_b[l] = temp;
                }
                break;
            }
            if (j == dim - 1)
            {
                *status = 1;
                return;
            }
        }
    }

    *status = 0;
    *scale = mat[i * 2 * dim + i];
}

__global__ void normalize_diagonal(double *mat, int i, int dim, double *scale)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    if (x < dim)
    {
        mat[i * 2 * dim + x] /= *scale;
        mat[i * 2 * dim + dim + x] /= *scale;
    }
}

__global__ void perform_elimination(int row_start, int row_end, int pivot_idx, double *mat, int col_start, int dim)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int i = x + row_start;

    if (i < row_end)
    {
        double scale = mat[i * 2 * dim + col_start];

        for (int col = col_start; col < 2 * dim; col++)
        {
            mat[i * 2 * dim + col] -= mat[2 * dim * pivot_idx + col] * scale;
        }
    }
}

void eliminate_col_from_pivot(int row_start, int row_end, int pivot_idx, double *mat, int col_start, int dim)
{
    int row_size = row_end - row_start;

    int gridsize = row_size / blocksize;

    if (row_size % blocksize != 0)
        gridsize++;

    perform_elimination<<<blocksize, gridsize>>>(row_start, row_end, pivot_idx, mat, col_start, dim);
}

int main(int argc, char *argv[])
{
    int dim = 0;

    scanf("%d", &dim);

    // case when dim = 3
    // core = 2
    int col_size = 2 * dim;
    int row_size = dim;

    // initialize matrix
    double *mat = (double *)malloc(row_size * col_size * sizeof(double));

    // scan matrix
    for (int i = 0; i < row_size; ++i)
    {
        for (int j = 0; j < dim; ++j)
        {
            scanf("%lf", &mat[i * col_size + j]);
        }
    }

    // Initializing Right-hand side to identity matrix
    for (int i = 0; i < dim; ++i)
    {
        for (int j = dim; j < col_size; ++j)
        {
            if (j == (i + dim))
            {
                mat[i * col_size + j] = 1;
            }
            else
            {
                mat[i * col_size + j] = 0;
            }
        }
    }

    // initialize cuda

    double *mat_d;
    hipMalloc(&mat_d, row_size * col_size * sizeof(double));
    hipMemcpy(mat_d, mat, row_size * col_size * sizeof(double), hipMemcpyHostToDevice);

    double *scale_d;
    hipMalloc(&scale_d, sizeof(double));

    int *status_d;
    int status;
    hipMalloc(&status_d, sizeof(int));

    struct timeval t1, t2;
    gettimeofday(&t1, NULL);

    for (int i = 0; i < dim; i++)
    {
        swap_zero_diagonal_and_get_scale<<<1, 1>>>(mat_d, i, dim, scale_d, status_d);

        hipMemcpy(&status, status_d, sizeof(int), hipMemcpyDeviceToHost);

        if (status != 0)
        {
            printf("No inverse exist\n");
            exit(0);
        }

        int gridsize = dim / blocksize;

        if (dim % blocksize != 0)
            gridsize++;

        normalize_diagonal<<<blocksize, gridsize>>>(mat_d, i, dim, scale_d);

        if (i == dim - 1)
            continue;

        eliminate_col_from_pivot(i + 1, dim, i, mat_d, i, dim);
    }

    for (int i = dim - 1; i >= 1; i--)
    {
        eliminate_col_from_pivot(0, i, i, mat_d, i, dim);
    }

    hipDeviceSynchronize();

    hipMemcpy(mat, mat_d, row_size * col_size * sizeof(double), hipMemcpyDeviceToHost);

    gettimeofday(&t2, NULL);
    double elapsed_time = t2.tv_sec - t1.tv_sec;

    printf("Elapsed time: %f seconds\n", elapsed_time);

    printf("\n=============RESULT FROM CUDA=============\n");
    printMat(mat, dim);

    free(mat);
    hipFree(mat_d);
    hipFree(scale_d);
    hipFree(status_d);

    return 0;
}